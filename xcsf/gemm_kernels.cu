
#include <hip/hip_runtime.h>
/*
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
                      
/**
 * @file gemm_kernels.cu
 * @author Richard Preen <rpreen@gmail.com>
 * @copyright The Authors.
 * @date 2020.
 * @brief CUDA general matrix multiplication.
 */ 
 
#ifdef GPU

#include <iostream>
#include "cuda.h"

__global__ void kernel_mm_multiply(const double *A, const double *B, double *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0;
    if(row < n && col < n) {
        for(int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * n + col];
        }
    }
    C[row * n + col] = sum;
}

__global__ void kernel_mv_multiply(const double *A, const double *B, double *C, int n)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0;
    if(row < n) {
        for(int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i];
        }
    }
    C[row] = sum;
}

extern "C" void gpu_mm_multiply(const double *A, const double *B, double *C, int n)
{
    int size = n*n;
    // allocate memory on the device
    double *d_a, *d_b, *d_c;
    CUDA_CALL( cudaMalloc((void **) &d_a, sizeof(double) * size) );
    CUDA_CALL( cudaMalloc((void **) &d_b, sizeof(double) * size) );
    CUDA_CALL( cudaMalloc((void **) &d_c, sizeof(double) * size) );

    // copy from host to device
    CUDA_CALL( cudaMemcpy(d_a, A, sizeof(double) * size, cudaMemcpyHostToDevice) );
    CUDA_CALL( cudaMemcpy(d_b, B, sizeof(double) * size, cudaMemcpyHostToDevice) );

    // run kernel on the GPU
    dim3 dimGrid(n,n);
    dim3 dimBlock(1,1);
    if(n > 65535) {
        dimBlock.x = sqrt(BLOCK_SIZE);
        dimBlock.y = dimBlock.x;
        dimGrid.x = (n % dimBlock.x == 0) ? n / dimBlock.x : (n / dimBlock.x) + 1;
    }
    kernel_mm_multiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

    // wait for GPU to finish
    CUDA_CALL( cudaDeviceSynchronize() );

    // copy result from device to host
    CUDA_CALL( cudaMemcpy(C, d_c, sizeof(double) * size, cudaMemcpyDeviceToHost) );

    // free memory
    CUDA_CALL( cudaFree(d_a) );
    CUDA_CALL( cudaFree(d_b) );
    CUDA_CALL( cudaFree(d_c) );
}

extern "C" void gpu_mv_multiply(const double *A, const double *B, double *C, int n)
{
    int size = n*n;
    // allocate memory on the device
    double *d_a, *d_b, *d_c;
    CUDA_CALL( cudaMalloc((void **) &d_a, sizeof(double) * size) );
    CUDA_CALL( cudaMalloc((void **) &d_b, sizeof(double) * n) );
    CUDA_CALL( cudaMalloc((void **) &d_c, sizeof(double) * n) );

    // copy from host to device
    CUDA_CALL( cudaMemcpy(d_a, A, sizeof(double) * size, cudaMemcpyHostToDevice) );
    CUDA_CALL( cudaMemcpy(d_b, B, sizeof(double) * n, cudaMemcpyHostToDevice) );

    // run kernel on the GPU
    dim3 dimGrid(n,n);
    dim3 dimBlock(1,1);
    if(n > 65535) {
        dimBlock.x = sqrt(BLOCK_SIZE);
        dimBlock.y = dimBlock.x;
        dimGrid.x = (n % dimBlock.x == 0) ? n / dimBlock.x : (n / dimBlock.x) + 1;
    }
    kernel_mv_multiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

    // wait for GPU to finish
    CUDA_CALL( cudaDeviceSynchronize() );

    // copy result from device to host
    CUDA_CALL( cudaMemcpy(C, d_c, sizeof(double) * n, cudaMemcpyDeviceToHost) );

    // free memory
    CUDA_CALL( cudaFree(d_a) );
    CUDA_CALL( cudaFree(d_b) );
    CUDA_CALL( cudaFree(d_c) );
}

#endif
