#include "hip/hip_runtime.h"
/*
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
                      
/**
 * @file cuda.cu
 * @author Richard Preen <rpreen@gmail.com>
 * @copyright The Authors.
 * @date 2020.
 * @brief General CUDA functions.
 */ 

#include <stdio.h>
#include <stdint.h>
#include "hip/hip_runtime.h"

static void cuda_printDeviceInfo(hipDeviceProp_t devProp);

int gpu_index = 0;

hipblasHandle_t blas_handle()
{
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    int i = cuda_get_device();
    if(!init[i]) {
        hipblasCreate(&handle[i]);
        init[i] = 1;
    }
    return handle[i];
}

dim3 cuda_gridsize(size_t n)
{
    size_t k = (n-1) / BLOCK_SIZE + 1;
    size_t x = k;
    size_t y = 1;
    if(x > 65535) {
        x = ceil(sqrt(k));
        y = (n-1) / (x*BLOCK_SIZE) + 1;
    }
    dim3 d;
    d.x = x;
    d.y = y;
    d.z = 1;
    return d;
}

__global__ void kernel_fill(int N, double *X, double ALPHA)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        X[i] = ALPHA;
    }
}

static void cuda_printDeviceInfo(hipDeviceProp_t devProp)
{
    printf("Revision number:               %d.%d\n", devProp.major, devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu MB\n",  devProp.totalGlobalMem / (1024 * 1024));
    printf("Total shared memory per block: %lu kB\n",  devProp.sharedMemPerBlock / 1024);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu MB\n",  devProp.memPitch / (1024 * 1024));
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    printf("Maximum dimensions of block:   %d %d %d\n", devProp.maxThreadsDim[0], devProp.maxThreadsDim[1], devProp.maxThreadsDim[2]);
    printf("Maximum dimensions of grid:    %d %d %d\n", devProp.maxGridSize[0], devProp.maxGridSize[1], devProp.maxGridSize[2]);
    printf("Clock rate:                    %d MHz\n",  devProp.clockRate / 1000);
    printf("Total constant memory:         %lu kB\n",  devProp.totalConstMem / 1024);
    printf("Texture alignment:             %lu B\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    printf("\n");
}

#ifdef __cplusplus
extern "C" {
#endif

void cuda_copy(int N, const double *src, double *dest)
{
    hipblasHandle_t handle = blas_handle();
    hipblasDcopy(handle, N, src, 1, dest, 1);
}

void cuda_fill(int N, double *X, double ALPHA) {
    kernel_fill<<<cuda_gridsize(N), BLOCK_SIZE>>>(N, X, ALPHA);
}

void cuda_set_device(int n)
{
    gpu_index = n;
    CUDA_CALL( hipSetDevice(n) );
}

int cuda_get_device()
{
    int n = 0;
    CUDA_CALL( hipGetDevice(&n) );
    return n;
}

double *cuda_make_array(const double *x, size_t n)
{
    double *x_gpu;
    size_t size = sizeof(double) * n;
    CUDA_CALL( hipMalloc((void **) &x_gpu, size) );
    if(x) {
        CUDA_CALL( hipMemcpy(x_gpu, x, size, hipMemcpyHostToDevice) );
    }
    else {
        CUDA_CALL( hipMemset(x_gpu, 0, size) );
    }
    return x_gpu;
}

void cuda_info()
{
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);
    for (int i = 0; i < devCount; i++) {
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        cuda_printDeviceInfo(devProp);
    }
}

#ifdef __cplusplus
}
#endif
