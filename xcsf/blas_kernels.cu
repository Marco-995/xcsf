#include "hip/hip_runtime.h"
/*
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
                      
/**
 * @file blas_kernels.cu
 * @author Richard Preen <rpreen@gmail.com>
 * @copyright The Authors.
 * @date 2020.
 * @brief CUDA basic linear algebra functions.
 */ 
 
#include <iostream>
#include <stdio.h>
#include "hipblas.h"
extern "C" {
#include "hip/hip_runtime.h"
}

hipblasHandle_t blas_handle()
{
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    int i = cuda_get_device();
    if(!init[i]) {
        hipblasCreate(&handle[i]);
        init[i] = 1;
    }
    return handle[i];
}

__device__ double atomic_Add(double *address, double val)
{
    unsigned long long int *address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void kernel_axpy(int N, double ALPHA, const double *X, int INCX, double *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        Y[i*INCY] += ALPHA * X[i*INCX];
    }
}

__global__ void kernel_scal(int N, double ALPHA, double *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        X[i*INCX] *= ALPHA;
    }
}

__global__ void kernel_sub(int N, double *A, double *B, double *C)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
        C[i] = A[i] - B[i];
    }
}

__global__ void kernel_dot(const double *A, const double *B, double *C, int N)
{
    __shared__ double cache;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    double temp = 0;
    cache = 0;
    __syncthreads();
    while (tid < N) {
        temp += A[tid] * B[tid];
        tid += blockDim.x * gridDim.x;
    }
    atomic_Add(&cache, temp);
    __syncthreads();
    if (cacheIndex == 0) {
        C[blockIdx.x] = cache;
    }
}

extern "C" void sub_gpu(int N, double *A, double *B, double *C, const hipStream_t *stream)
{
    kernel_sub<<<cuda_gridsize(N), BLOCK_SIZE, 0, *stream>>>(N, A, B, C);
}

extern "C" void scal_gpu(int N, double ALPHA, double *X, int INCX, const hipStream_t *stream)
{
    kernel_scal<<<cuda_gridsize(N), BLOCK_SIZE, 0, *stream>>>(N, ALPHA, X, INCX);
}

extern "C" void axpy_gpu(int N, double ALPHA, const double *X, int INCX, double *Y, int INCY,
        const hipStream_t *stream)
{
    kernel_axpy<<<cuda_gridsize(N), BLOCK_SIZE, 0, *stream>>>(N, ALPHA, X, INCX, Y, INCY);
}

extern "C" void dot_gpu(int N, const double *A, const double *B, double *C,
        const hipStream_t *stream)
{
    kernel_dot<<<cuda_gridsize(N), BLOCK_SIZE, 0, *stream>>>(A, B, C, N);
}

extern "C" void gemm_gpu(int TA, int TB, int M, int N, int K, double ALPHA,
        const double *A, int lda,
        const double *B, int ldb,
        double BETA,
        double *C, int ldc,
        const hipStream_t *stream)
{
    hipblasHandle_t handle = blas_handle();
    hipblasDgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
            (TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B, ldb, A, lda, &BETA, C, ldc);
}
